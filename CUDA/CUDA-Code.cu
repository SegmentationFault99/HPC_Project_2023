
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define DEFAULT_WIDTH 20
#define DEFAULT_HEIGTH 20
#define DEFAULT_ITERATIONS 20
#define DEFAULT_THREADS 32

//computes the difference between two times, in milliseconds
double compute_time_interval(struct timeval start_time, struct timeval end_time)
{
	return (double)((end_time.tv_sec - start_time.tv_sec)*1000000 + (end_time.tv_usec - start_time.tv_usec)) / 1000;
}

//saves the results passed as a parameter in a result file
void write_to_file(int width, int heigth, int threads, double total_time)
{
	FILE * f;
	f = fopen("CUDA-Results.csv", "a");

	fprintf(f, "%d,%d,%d,%f\n", width, heigth, threads, total_time);
	fflush(f);
	fclose(f);
}

//swaps the content of a matrix with the content of another one (by swapping their pointers)
void swap_matrix(unsigned int **old, unsigned int **new_)
{
	unsigned int *temp = *old;
	*old = *new_;
	*new_ = temp;
}

//computes a single round of the game
__global__ void one_round(unsigned int *grid, unsigned int *new_grid, int heigth, int width)
{
	const int grid_size = heigth * width;
	const int position = blockIdx.x * blockDim.x + threadIdx.x;

	if (position >= grid_size)
		return;

	//computing position of the current cell
	int column_index = position % width;
	int row_index = position - column_index;

	//computing positions of the cells surrounding the current one
	int left = (column_index + width - 1) % width;
	int right = (column_index + 1) % width;
	int top = (row_index + grid_size - width) % grid_size;
	int bottom = (row_index + width) % grid_size;

	//the computed indexes are combined to locate all of the cells surrounding the current one, their values are summed up in order to find the number of active neighbours
	int active_neighbours = grid[left + top] + grid[column_index + top] + grid[right + top] + grid[left + row_index] + grid[right + row_index] + grid[left + bottom] + grid[column_index + bottom] + grid[right + bottom];

	//update of the value of the cell according to the rules of the game
	if (grid[column_index + row_index] == 1 && (active_neighbours < 2 || active_neighbours > 3))
		new_grid[column_index + row_index] = 0;
	else if ((grid[column_index + row_index] == 1 && (active_neighbours == 2 || active_neighbours == 3)) || (grid[column_index + row_index] == 0 && active_neighbours == 3))
		new_grid[column_index + row_index] = 1;
	else
		new_grid[column_index + row_index] = grid[column_index + row_index];
}

//main auxiliary function, initializes the grid situation and monitors its evolution
void play(int width, int heigth, int num_threads)
{
	int i, j;
	struct timeval start_time, end_time;
	double total_time = 0.0;
	unsigned int *grid = (unsigned int *) malloc(heigth *width* sizeof(unsigned int));

	//initialization
	for (i = 0; i < heigth; i++)
		for (j = 0; j < width; j++)
			grid[i *width + j] = rand() % 10 >= 7 ? 1 : 0;

	size_t grid_size = heigth *width* sizeof(unsigned int);
	unsigned int *cuda_grid;
	unsigned int *cuda_new_grid;
	hipMalloc((void **) &cuda_grid, grid_size);
	hipMalloc((void **) &cuda_new_grid, grid_size);
	hipMemcpy(cuda_grid, grid, grid_size, hipMemcpyHostToDevice);
	dim3 threads(num_threads);
	dim3 chunks((int)(heigth *width + threads.x - 1) / threads.x);

	for (i = 0; i < DEFAULT_ITERATIONS; i++)
	{
		gettimeofday(&start_time, NULL);

		//computation of the game evolution in this iteration
		one_round <<<chunks, threads>>> (cuda_grid, cuda_new_grid, heigth, width);

		hipDeviceSynchronize();
		gettimeofday(&end_time, NULL);

		swap_matrix(&cuda_grid, &cuda_new_grid);
		total_time += (double) compute_time_interval(start_time, end_time);
	}

	write_to_file(width, heigth, num_threads, total_time);
	hipFree(cuda_grid);
	hipFree(cuda_new_grid);
	free(grid);
}

int main(int argc, char **argv)
{
	//width, heigth and chunk size are initialized with their default values
	int width = DEFAULT_WIDTH,
		heigth = DEFAULT_HEIGTH,
		threads = DEFAULT_THREADS;

	if (argc != 4)
	{
		printf("Invalid number of parameters");
		return 0;
	}

	//if the program was launched with correct custom parameters, the default values are replaced with them
	if (atoi(argv[1]) > DEFAULT_WIDTH)
		width = atoi(argv[1]);

	if (atoi(argv[2]) > DEFAULT_HEIGTH)
		heigth = atoi(argv[2]);

	if (atoi(argv[3]) > DEFAULT_THREADS)
		threads = atoi(argv[3]);

	play(width, heigth, threads);
}